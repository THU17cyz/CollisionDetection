#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

extern "C"
{

    void cudaInit(int argc, char **argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char **)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

	void copyArrayFromDevice(void *host, const void *device, int size)
	{
		checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
		//hipMemcpyFromSymbol(host, HIP_SYMBOL(device), size);
	}

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void update_dynamics(float *pos, float *velo, float *radius, float elapse, uint sphere_num)
    {
        /*thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_vel4((float4 *)vel);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
            integrate_functor(deltaTime));*/
		uint numThreads, numBlocks;
		computeGridSize(sphere_num, 256, numBlocks, numThreads);
		update_dynamics<<< numBlocks, numThreads >>>(
			(float3 *) pos, 
			(float3 *) velo, 
			radius, 
			elapse, 
			sphere_num);
		getLastCudaError("Kernel execution failed");
    }

    void calcHash(uint  *hash, uint  *index, float *pos, uint sphere_num)
    {
        uint numThreads, numBlocks;
        computeGridSize(sphere_num, 256, numBlocks, numThreads);
        calcHashD<<< numBlocks, numThreads >>>(
			hash,
			index,
			(float3 *) pos,
			sphere_num);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *pos_sorted,
                                     float *velo_sorted,
                                     uint  *hash,
                                     uint  *index,
                                     float *oldPos,
                                     float *oldVel,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));
        reorderDataAndFindCellStartD<<< numBlocks, numThreads >>>(
            cellStart,
            cellEnd,
            (float3 *) pos_sorted,
            (float3 *)velo_sorted,
            hash,
            index,
            (float3 *) oldPos,
            (float3 *) oldVel,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

    }

    void collide(float *newVel,
                 float *pos_sorted,
                 float *velo_sorted,
		         float *radius,
		float *mass,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells)
    {

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float3 *)newVel,
                                              (float3 *)pos_sorted,
                                              (float3 *)velo_sorted,
			radius, mass,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

    }


    void radixSortByHash(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }

}   // extern "C"
